#include <hip/hip_runtime.h>
#include <stdio.h>


typedef struct {
    int id;
    unsigned int flags;
    int active;
} primaryCtxState;



int main (int argc, char * argv[]) {

    int n_dev;
    hipGetDeviceCount(& n_dev);

    printf("%d devices are available\n", n_dev);

    if (n_dev <= 0) return 1;


    primaryCtxState state[n_dev];


    printf("Checking state of primary context -- before first CUDA runtime API call\n");


    for (int i=0; i<n_dev; i++) {
        state[i].id = i;
        hipDevicePrimaryCtxGetState(
                state[i].id,
                & state[i].flags, & state[i].active
            );
    }

    for (int i=0; i<n_dev; i++) {
        printf(
                "Device %d state: flags=%d, active=%d\n",
                state[i].id, state[i].flags, state[i].active
            );
    }


    printf("Checking state of primary context -- after a hipMalloc on each device\n");

    hipCtx_t context[n_dev];
    hipDevice_t  device[n_dev];

    for (int i=0; i<n_dev; i++) {
        hipDeviceGet(& device[i], i);
        hipCtxCreate(& context[i], 0, device[i]);
        void * p;
        hipMalloc(& p, 1);
    }

    for (int i=0; i<n_dev; i++) {
        hipDevicePrimaryCtxGetState(
                state[i].id,
                & state[i].flags, & state[i].active
            );
    }

    for (int i=0; i<n_dev; i++) {
        printf(
                "Device %d state: flags=%d, active=%d\n",
                state[i].id, state[i].flags, state[i].active
            );
    }


    return 0;
}
