#include <hip/hip_runtime.h>
#include <stdio.h>
#include <util.h>



int main (int argc, char * argv[]) {

    int n_dev;
    hipGetDeviceCount(& n_dev);

    printf("%d devices are available\n", n_dev);
    if (n_dev <= 0) return 1;


    printf("Checking state of primary context -- before first CUDA runtime API call\n");
    check_primary_ctx(n_dev);


    printf("Checking state of primary context -- after a hipMalloc on each device\n");

    hipCtx_t context[n_dev];
    hipDevice_t  device[n_dev];

    for (int i=0; i<n_dev; i++) {
        hipDeviceGet(& device[i], i);
        hipCtxCreate(& context[i], 0, device[i]);
        void * p;
        hipMalloc(& p, 1);
    }

    check_primary_ctx(n_dev);

    return 0;
}
