#include <hip/hip_runtime.h>
#include <stdio.h>
#include <util.h>


int main (int argc, char * argv[]) {

    int n_dev;
    hipGetDeviceCount(& n_dev);

    printf("%d devices are available\n", n_dev);
    if (n_dev <= 0) return 1;


    printf("Checking state of primary context -- before first CUDA runtime API call\n");
    check_primary_ctx(n_dev);


    printf("Checking state of primary context -- after a hipSetDevice on device 0 only\n");

    hipSetDevice(0);

    check_primary_ctx(n_dev);

    return 0;
}
