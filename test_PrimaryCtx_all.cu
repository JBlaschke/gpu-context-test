#include <hip/hip_runtime.h>
#include <stdio.h>
#include <util.h>



int main (int argc, char * argv[]) {

    int n_dev;
    hipGetDeviceCount(& n_dev);

    printf("%d devices are available\n", n_dev);
    if (n_dev <= 0) return 1;


    printf("Checking state of primary context -- before first CUDA runtime API call\n");
    check_primary_ctx(n_dev);


    printf("Checking state of primary context -- after a hipMalloc on each device\n");

    for (int i=0; i<n_dev; i++) {
        hipSetDevice(i);
        void * p;
        hipMalloc(& p, 1);
    }

    hipSetDevice(0);

    check_primary_ctx(n_dev);

    return 0;
}
