#include <hip/hip_runtime.h>
#include <stdio.h>
#include <util.h>



int main (int argc, char * argv[]) {

    int n_dev;
    hipGetDeviceCount(& n_dev);

    printf("%d devices are available\n", n_dev);
    if (n_dev <= 0) return 1;


    printf("Checking state of primary context -- before first CUDA runtime API call\n");
    check_primary_ctx(n_dev);


    printf("Checking state of primary context -- after a hipMalloc on each device\n");

    hipCtx_t context;
    hipDevice_t  device;

    hipDeviceGet(& device, 0);
    hipCtxCreate(& context, 0, device);
    void * p;
    hipMalloc(& p, 1);

    for (int i=1; i<n_dev; i++) {
        hipSetDevice(i);
        void * p;
        hipMalloc(& p, 1);
    }

    hipSetDevice(0);

    check_primary_ctx(n_dev);

    return 0;
}
