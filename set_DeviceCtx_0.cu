#include <hip/hip_runtime.h>
#include <stdio.h>


typedef struct {
    int id;
    unsigned int flags;
    int active;
} primaryCtxState;



int main (int argc, char * argv[]) {

    int n_dev;
    hipGetDeviceCount(& n_dev);

    printf("%d devices are available\n", n_dev);

    if (n_dev <= 0) return 1;


    primaryCtxState state[n_dev];


    printf("Checking state of primary context -- before first CUDA runtime API call\n");


    for (int i=0; i<n_dev; i++) {
        state[i].id = i;
        hipDevicePrimaryCtxGetState(
                state[i].id,
                & state[i].flags, & state[i].active
            );
    }

    for (int i=0; i<n_dev; i++) {
        printf(
                "Device %d state: flags=%d, active=%d\n",
                state[i].id, state[i].flags, state[i].active
            );
    }


    printf("Checking state of primary context -- after a hipMalloc on each device\n");

    hipCtx_t context;
    hipDevice_t  device;

    hipDeviceGet(& device, 0);
    hipCtxCreate(& context, 0, device);
    void * p;
    hipMalloc(& p, 1);

    for (int i=1; i<n_dev; i++) {
        hipSetDevice(i);
        void * p;
        hipMalloc(& p, 1);
    }

    hipSetDevice(0);

    for (int i=0; i<n_dev; i++) {
        hipDevicePrimaryCtxGetState(
                state[i].id,
                & state[i].flags, & state[i].active
            );
    }

    for (int i=0; i<n_dev; i++) {
        printf(
                "Device %d state: flags=%d, active=%d\n",
                state[i].id, state[i].flags, state[i].active
            );
    }


    return 0;
}
