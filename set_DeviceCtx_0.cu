#include <hip/hip_runtime.h>
#include <stdio.h>
#include <util.h>



int main (int argc, char * argv[]) {

    int n_dev;
    hipGetDeviceCount(& n_dev);

    printf("%d devices are available\n", n_dev);
    if (n_dev <= 0) return 1;


    printf("Checking state of primary context -- before first CUDA runtime API call\n");
    check_primary_ctx(n_dev);


    printf("Checking state of primary context -- after saxpy on each device sequentially\n");

    int M = 1000;
    int * h_x = (int *) malloc(M*sizeof(int));
    int * h_y = (int *) calloc(M, sizeof(int));
    for (int i=0; i<M; i++) h_x[i] = 10;

    hipCtx_t context;
    hipDevice_t  device;

    hipDeviceGet(& device, 0);
    hipCtxCreate(& context, 0, device);
    hipCtxPushCurrent(context);

    printf("NOTE: NOT calling hipSetDevice on device 0\n");
    for (int i=0; i<n_dev; i++) {
        if (i>0) hipSetDevice(i);

        int * d_x, * d_y;
        api_malloc(& d_x, M);
        api_malloc(& d_y, M);
        api_HtoD(h_x, d_x, M);
        api_HtoD(h_y, d_y, M);
        saxpy_int<<<(M+255)/256, 256>>>(M, 1, d_x, d_y);
        api_DtoH(h_x, d_x, M);
        api_DtoH(h_y, d_y, M);
        hipFree(d_x);
        hipFree(d_y);
        printf("Device %d work result: %d\n", i, h_y[0]);
    }

    check_primary_ctx(n_dev);

    printf("NOW calling hipSetDevice on device 0\n");
    hipSetDevice(0);
    int * d_x, * d_y;
    api_malloc(& d_x, M);
    api_malloc(& d_y, M);
    api_HtoD(h_x, d_x, M);
    api_HtoD(h_y, d_y, M);
    saxpy_int<<<(M+255)/256, 256>>>(M, 1, d_x, d_y);
    api_DtoH(h_x, d_x, M);
    api_DtoH(h_y, d_y, M);
    hipFree(d_x);
    hipFree(d_y);
    printf("Device %d work result: %d\n", 0, h_y[0]);
    check_primary_ctx(n_dev);

    free(h_x);
    free(h_y);
    return 0;
}
