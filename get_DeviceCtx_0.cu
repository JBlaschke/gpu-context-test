#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>


typedef struct {
    int id;
    unsigned int flags;
    int active;
} primaryCtxState;



int main (int argc, char * argv[]) {

    int n_dev;
    hipGetDeviceCount(& n_dev);

    printf("%d devices are available\n", n_dev);

    if (n_dev <= 0) return 1;


    primaryCtxState state[n_dev];


    printf("Checking state of primary context -- before first CUDA runtime API call\n");


    for (int i=0; i<n_dev; i++) {
        state[i].id = i;
        hipDevicePrimaryCtxGetState(
                state[i].id,
                & state[i].flags, & state[i].active
            );
    }

    for (int i=0; i<n_dev; i++) {
        printf(
                "Device %d state: flags=%d, active=%d\n",
                state[i].id, state[i].flags, state[i].active
            );
    }


    printf("Checking state of primary context -- after a hipMalloc on each device\n");

    int cu_dev = 1;

    hipCtx_t context;
    hipDevice_t  device;

    hipDeviceGet(& device, cu_dev);
    hipCtxCreate(& context, 0, device);

    int ierr = hipCtxGetDevice(& device);
    printf("%d, %d\n", device, ierr);

    int dev;
    hipGetDevice(& dev);
    printf("cuda api dev=%d\n", dev);

    for (int i=0; i<n_dev; i++) {
        if (i != cu_dev) hipSetDevice(i);
        int * d_p;
        hipMalloc(& d_p, sizeof(int));
        int payload = 10;
        int * p_payload = & payload;
        hipMemcpy(d_p, p_payload, sizeof(int), hipMemcpyHostToDevice);
        int r_payload = 0;
        int * p_r_payload = & r_payload;
        hipMemcpy(p_r_payload, d_p, sizeof(int), hipMemcpyDeviceToHost);
        printf("Device %d round-trip int=%d\n", i, r_payload);
    }

    hipSetDevice(0);

    for (int i=0; i<n_dev; i++) {
        hipDevicePrimaryCtxGetState(
                state[i].id,
                & state[i].flags, & state[i].active
            );
    }

    for (int i=0; i<n_dev; i++) {
        printf(
                "Device %d state: flags=%d, active=%d\n",
                state[i].id, state[i].flags, state[i].active
            );
    }


    return 0;
}
