#include "hip/hip_runtime.h"
#include <util.h>
#include <hip/hip_runtime.h>
#include <stdio.h>



void check_primary_ctx(int n_dev) {

    primaryCtxState state[n_dev];

    for (int i=0; i<n_dev; i++) {
        state[i].id = i;
        hipDevicePrimaryCtxGetState(
                state[i].id,
                & state[i].flags, & state[i].active
            );
    }

    for (int i=0; i<n_dev; i++) {
        printf(
                "Device %d state: flags=%d, active=%d\n",
                state[i].id, state[i].flags, state[i].active
            );
    }
}


void api_malloc(int ** d_p, int M) {
    hipMalloc(d_p, sizeof(int)*M);
}



void api_HtoD(int * h_p, int * d_p, int M) {
    hipMemcpy(d_p, h_p, sizeof(int)*M, hipMemcpyHostToDevice);
}



void api_DtoH(int * h_p, int * d_p, int M) {
    hipMemcpy(h_p, d_p, sizeof(int)*M, hipMemcpyDeviceToHost);
}



__global__
void saxpy_int(int n, int a, int * x, int * y) {

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) y[i] = a*x[i] + y[i];
}
