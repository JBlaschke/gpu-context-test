#include "hip/hip_runtime.h"
#include <util.h>
#include <hip/hip_runtime.h>
#include <stdio.h>



void check_primary_ctx(int n_dev) {

    primaryCtxState state[n_dev];

    for (int i=0; i<n_dev; i++) {
        state[i].id = i;
        hipDevicePrimaryCtxGetState(
                state[i].id,
                & state[i].flags, & state[i].active
            );
    }

    for (int i=0; i<n_dev; i++) {
        printf(
                "Device %d state: flags=%d, active=%d\n",
                state[i].id, state[i].flags, state[i].active
            );
    }
}



hipError_t get_current_device(int * i_dev, int * is_primary, int * is_clean) {

    hipError_t cuda_err;
    hipError_t ierr;

    // GET current device bound to this thread (this will work for _both_ the
    // cuda runtim API and the cuda driver)
    cuda_err = hipGetDevice(i_dev);
    if (cuda_err != hipSuccess) {
        * i_dev      = -1;
        * is_primary = -1;
        * is_clean   = -1;
        return hipErrorInvalidContext;
    }

    // GET the state of the primary context
    unsigned int flags;
    int active;
    hipDevicePrimaryCtxGetState(* i_dev, & flags, & active);
    if (active == 1){
        * is_primary = 1;
        * is_clean   = 0;
        return hipSuccess;
    }

    // The PRIMARY CONTEXT could be the only context on the device bound to
    // this thread -- and just not be active because nothing has been called
    // it, or another context is the current context

    hipDevice_t device;

    ierr = hipCtxGetDevice(& device);
    if (ierr == hipErrorInvalidContext) {
        * is_primary = 1;
        * is_clean   = 1;
    } else if (ierr != hipSuccess) {
        return ierr;
    }

    // There is defintely a device bound to this thread -- so there MUST be a
    // current context. The only thing to figure out now is if this context is
    // the primary context (which just hasn't been used yet), or another.

    hipCtx_t context;

    ierr = hipCtxGetCurrent(& context);
    if (ierr != hipSuccess)
        return ierr;

    hipCtx_t primary_context;
    // NOTE: this will make the primary context active -- we need to release it
    // again below:
    ierr = hipDevicePrimaryCtxRetain(& primary_context, device);
    if (ierr != hipSuccess)
        return ierr;
    // Restore the pre-existing context by de-activating the primary context
    // that cudaPrimaryCtxRetain activated
    ierr = hipDevicePrimaryCtxRelease(device);
    if (ierr != hipSuccess)
        return ierr;

    if (primary_context == context) {
        * is_primary = 1;
        * is_clean   = 0;

        return hipSuccess;
    }

    * is_primary = 0;
    * is_clean   = 0;

    return hipSuccess;
}



void api_malloc(int ** d_p, int M) {
    hipMalloc(d_p, sizeof(int)*M);
}



void api_HtoD(int * h_p, int * d_p, int M) {
    hipMemcpy(d_p, h_p, sizeof(int)*M, hipMemcpyHostToDevice);
}



void api_DtoH(int * h_p, int * d_p, int M) {
    hipMemcpy(h_p, d_p, sizeof(int)*M, hipMemcpyDeviceToHost);
}



__global__
void saxpy_int(int n, int a, int * x, int * y) {

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) y[i] = a*x[i] + y[i];
}
