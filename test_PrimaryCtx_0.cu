#include <hip/hip_runtime.h>
#include <stdio.h>
#include <util.h>


int main (int argc, char * argv[]) {

    int n_dev;
    hipGetDeviceCount(& n_dev);

    printf("%d devices are available\n", n_dev);
    if (n_dev <= 0) return 1;


    printf("Checking state of primary context -- before first CUDA runtime API call\n");
    check_primary_ctx(n_dev);


    printf("Checking state of primary context -- after a hipMalloc on device 0 only\n");

    int M = 1000;
    int * h_x = (int *) malloc(M*sizeof(int));
    int * h_y = (int *) calloc(M, sizeof(int));
    for (int i=0; i<M; i++) h_x[i] = 10;
 
    hipSetDevice(0);
    int * d_x, * d_y;
    api_malloc(& d_x, M);
    api_malloc(& d_y, M);
    api_HtoD(h_x, d_x, M);
    api_HtoD(h_y, d_y, M);
    saxpy_int<<<(M+255)/256, 256>>>(M, 1, d_x, d_y);
    api_DtoH(h_x, d_x, M);
    api_DtoH(h_y, d_y, M);
    hipFree(d_x);
    hipFree(d_y);
    printf("Device %d work result: %d\n", 0, h_y[0]);

    check_primary_ctx(n_dev);

    free(h_x);
    free(h_y);
    return 0;
}
